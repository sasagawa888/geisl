#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define IDX3C(c,i,j,in_h,in_w) ((c)*((in_h)*(in_w)) + (i)*(in_w) +(j))
#define IDX4C(n,c,i,j,in_c,in_h,in_w) ((n)*((in_c)*(in_h)*(in_w)) + (c)*((in_h)*(in_w)) + (i)*(in_w) +(j))
#define IDX5C(t,n,c,i,j,in_n,in_c,in_h,in_w) ((t)*((in_n)*(in_c)*(in_h)*(in_w)) + (n)*((in_c)*(in_h)*(in_w)) + (c)*((in_h)*(in_w)) + (i)*(in_w) +(j))
#define SIGMOID(x)  (1 / (1+exp(-1*x)))

#define CHECK(call)                                   \
{                                                     \
    const hipError_t error = call;                   \
    if (error != hipSuccess)                         \
    {                                                 \
        printf("cuda error %d",(int)error);           \
    }                                                 \
}

extern "C" void cuda_add(float *a, float *b, float *c, int n);
extern "C" void cuda_sub(float *a, float *b, float *c, int n);
extern "C" void cuda_smult(float s, int n, float *a, float *b);
extern "C" void cuda_pooling(int in_n, int in_c, int in_h, int in_w, float *a, float *b, float *c, int st_h, int st_w);
extern "C" void cuda_unpooling(int in_n, int in_c, int in_h, int in_w, float *a, float *b, float *c,int st_h, int st_w);
extern "C" void cuda_convolute(int in_n, int in_c, int in_h, int in_w, int filt_n, int filt_c, int filt_h, int filt_w, float *a, float *b, float *c,
    int st_h, int st_w, int pad);
extern "C" void cuda_deconvolute1(int in_n, int in_c, int in_h, int in_w, int filt_n, int filt_c, int filt_h, int filt_w,
    float *a, float *b, float *c, int st_h, int st_w, int pad);
extern "C" void cuda_deconvolute2(int in_n, int in_c, int in_h, int in_w, int filt_n, int filt_c, int filt_h, int filt_w,
        float *a, float *b, float *c, int st_h, int st_w, int pad);
extern "C" void cuda_gradfilter1(int in_n, int in_c, int in_h, int in_w, int filt_n, int filt_c, int filt_h, int filt_w, 
    int loss_c, int loss_h, int loss_w, float *a, float *b, float *d, int st_h, int st_w, int pad);
extern "C" void cuda_gradfilter2(int in_n, int in_c, int in_h, int in_w, int filt_n, int filt_c, int filt_h, int filt_w, 
    int loss_c, int loss_h, int loss_w, float *a, float *b, float *d, int st_h, int st_w, int pad);
extern "C" void cuda_emult(int n, float *a, float *b,float *c);
extern "C" void cuda_full(int in_n, int in_c, int in_h, int in_w, float *a, float *b);
extern "C" void cuda_unfull(int in_n, int in_c, int in_h, int in_w, float *a, float *b);
extern "C" void cuda_activate_sigmoid(int n, float *a, float *b);
extern "C" void cuda_activate_tanh(int n, float *a, float *b);
extern "C" void cuda_activate_relu(int n, float *a, float *b);
extern "C" void cuda_activate_softmax(int r1, int c1, float *a, float *b);
extern "C" void cuda_differ_sigmoid(int n, float *a, float *b, float *c);
extern "C" void cuda_differ_tanh(int n, float *a, float *b, float *c);
extern "C" void cuda_differ_relu(int n, float *a, float *b, float *c);
extern "C" void cuda_dropout(int n, float dropout_rate, float *a);
extern "C" void cuda_sgd(int n, float *a, float *b, float *c, float lr);
extern "C" void cuda_momentum(int n, float *a, float *b, float *c, float *d, float *e, float lr);
extern "C" void cuda_adagrad(int n, float *a, float *b, float *c, float *d, float *e, float lr);
extern "C" void cuda_rms(int n, float *a, float *b, float *c, float *d, float *e, float  lr);
extern "C" void cuda_adam(int n, float *a, float *b, float *c, float *d, float *e, float *f, float *g, float lr);

__global__ void add1_kernel(float *a, float *b, float *c, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
}

void cuda_add(float *a, float *b, float *c, int n){
    float *dev_a, *dev_b, *dev_c;

	
    // Allocate for GPU
	CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
	CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

	add1_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

	// copy to host c from GPU dev_c
	CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));
    
    // free 
    hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

}




__global__ void sub1_kernel(float *a, float *b, float *c, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n)
	{
		c[tid] = a[tid] - b[tid];
		tid += blockDim.x * gridDim.x;
	}
}


void cuda_sub(float *a, float *b, float *c, int n){
    float *dev_a, *dev_b, *dev_c;

	
    // Allocate for GPU
	hipMalloc((void**)&dev_a, n * sizeof(float));
	hipMalloc((void**)&dev_b, n * sizeof(float));
	hipMalloc((void**)&dev_c, n * sizeof(float));


    // copy from host a,b to GPU dev_a, dev_b
	hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice);

	sub1_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

	// copy to host c from GPU dev_c
	hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost);
    
    // free 
    hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

}




__global__ void smult_kernel(float d, float *a, float *b, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n)
    {
        b[tid] = d * a[tid];
        tid += blockDim.x * gridDim.x;
    }
}



void cuda_smult(float s, int n, float *a, float *b){
    float *dev_a, *dev_b;

    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

    smult_kernel << <128, 128 >> >(s,dev_a, dev_b, n);

    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(b, dev_b, n * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
    hipFree(dev_b);

}


__global__ void pooling_kernel(float *a, float *b, float *c, int st_h, int st_w, int in_c, int in_h, int in_w)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int n1,c1,h1,w1,h2,w2,in_h2,in_w2,start_h1,end_h1,start_w1,end_w1,max_h,max_w;
    float max,fmax_h,fmax_w; 
    n1 = bid;
    c1 = tid;
    in_h2 = in_h / st_h;
    in_w2 = in_w / st_w;
    
    for(w2=0;w2<in_w2;w2++){
        for(h2=0;h2<in_h2;h2++){
            max = -999999999.0;
            start_h1 = st_h*h2;
            end_h1 = st_h*(h2+1);
            start_w1 = st_w*w2;
            end_w1 = st_w*(w2+1);
            for(h1=start_h1;h1<end_h1;h1++){
                for(w1=start_w1;w1<end_w1;w1++){
                    if(a[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)] >= max){
                        max = a[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)];
                        max_h = h1;
                        max_w = w1;
                    }
                }
            }
            b[IDX4C(n1,c1,h2,w2,in_c,in_h2,in_w2)] = max;
            fmax_h = (float)max_h;
            fmax_w = (float)max_w;
            c[IDX4C(n1,c1,h2,w2,in_c,in_h2,in_w2)] = fmax_h * 1000.0 + fmax_w; 
        }
    }
    
}
  
  /*
  1st arg in_n of tensor
  2nd arg in_c of tensor
  3rd arg in_h of tensor
  4th arg in_w of tensor
  5th arg input tensor
  6th arg output tensor (forward result)
  7th arg output tensor (backward result)
  8th arg stride hight
  9th arg stride width

  return list [ts1,ts2]
  ts1 is result data for forward
  ts2 is result data dor backward. this is sparse matrix 
  e.g. 
  |0.1,0.2,0.3,0.4|
  |0.5,0.6,0.7,0.8|
  |0.9,1.0,1.1,1.2|
  |1.3,1.4,1.5,1.6|
  
  ts1
  |0.6,0.8|
  |1.4,1.6|

  ts2
  each element is  row*1000+col
  |1.0*1000+1.0,1.0*1000*3.0|
  |3.0*1000+1.0,3.0*1000+3.0|
  
  */
  
  void cuda_pooling(int in_n, int in_c, int in_h, int in_w, float *a, float *b, float *c, int st_h, int st_w){
	  int n1, n2;
	  float *dev_a, *dev_b, *dev_c;
	
	  
	  n1 = in_n * in_c * in_h * in_w;
	  n2 = in_n * in_c * (in_h / st_h) * (in_w / st_w);
	  
	  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_c, n2 * sizeof(float)));
	
	  // copy from host a to GPU dev_a
	  CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
	  
	  dim3 blocks(in_n,1,1);
	  dim3 threads(in_c,1,1);
	  pooling_kernel <<<blocks, threads>>>(dev_a, dev_b, dev_c, st_h, st_w, in_c, in_h, in_w);
	
	  // copy to host b,c from GPU dev_b,dev_c
	  CHECK(hipMemcpy(b, dev_b, n2 * sizeof(float), hipMemcpyDeviceToHost));
	  CHECK(hipMemcpy(c, dev_c, n2 * sizeof(float), hipMemcpyDeviceToHost));
		
  
	  // return forward data and backward data {b_bin,c_bin} 
	  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
	  hipFree(dev_c);

  }
  
  
  __global__ void unpooling_kernel(float *a, float *b, float *c, int st_h, int st_w, int in_c, int in_h, int in_w)
  {
	  int bid = blockIdx.x;
	  int tid = threadIdx.x;
	  int n1,c1,h1,w1,h2,w2,start_h1,end_h1,start_w1,end_w1,max_h,max_w,in_h1,in_w1;
	  float loss,elt;
  
	  n1 = bid;
	  c1 = tid;
	  in_h1 = in_h * st_h;
	  in_w1 = in_w * st_w;
	  for(h2=0;h2<in_h;h2++){
		  for(w2=0;w2<in_w;w2++){
			  start_h1 = st_h*h2;
			  end_h1 = st_h*(h2+1);
			  start_w1 = st_w*w2;
			  end_w1 = st_w*(w2+1);
			  elt = a[IDX4C(n1,c1,h2,w2,in_c,in_h,in_w)];
			  loss = b[IDX4C(n1,c1,h2,w2,in_c,in_h,in_w)];
			  max_h = (int) floor(elt / 1000.0);
			  max_w = (int) fmodf(elt,1000.0);
			  for(h1=start_h1;h1<end_h1;h1++){
				  for(w1=start_w1;w1<end_w1;w1++){
					  if(h1 == max_h && w1 == max_w){
						  c[IDX4C(n1,c1,h1,w1,in_c,in_h1,in_w1)] = loss;
					  }
					  else{
						  c[IDX4C(n1,c1,h1,w1,in_c,in_h1,in_w1)] = 0.0;
					  }
				  }
			  }
		  }
	  }
		  
  }
	
  /*
  1st arg in_n of sparse-tensor
  2nd arg in_c of sparse-tensor
  3rd arg in_h of sparse-tensor
  4th arg in_w of sparse-tensor
  5th arg input sparse-tensor
  6th arg input loss-tensor
  7th arg output tensor(backward)
  8th arg stride hight
  9th arg stride width
  
  return gradiate tensor
  e.g.
  ts1 index-tensor
	each element is  row*1000+col
	|1.0*1000+1.0,1.0*1000*3.0|
	|3.0*1000+1.0,3.0*1000+3.0|
  ts2 loss-tensor
	|0.1,0.2|
	|0.3,0.4|
  
  return
	|0.0,0.0,0.0,0.0|
	|0.0,0.1,0.0,0.2|
	|0.0,0.0,0.0,0.0|
	|0.0,3.4,0.0,0.4|
  
  */
  
  void cuda_unpooling(int in_n, int in_c, int in_h, int in_w, float *a, float *b, float *c,int st_h, int st_w){
	  int n1, n2;
	  float *dev_a, *dev_b, *dev_c;
	
	  n1 = in_n * in_c * in_h * in_w;
	  n2 = in_n * in_c * (in_h * st_h) * (in_w * st_w);
	  
		
	  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n1 * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_c, n2 * sizeof(float)));
  
	
	  // copy from host a,b to GPU dev_a, dev_b
	  CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_b, b, n1 * sizeof(float), hipMemcpyHostToDevice));

	  
	  dim3 blocks(in_n,1,1);
	  dim3 threads(in_c,1,1);
	  unpooling_kernel <<<blocks, threads>>>(dev_a, dev_b, dev_c, st_h, st_w, in_c, in_h, in_w);
	
	  // copy to host d from GPU dev_d
	  CHECK(hipMemcpy(c, dev_c, n2 * sizeof(float), hipMemcpyDeviceToHost));
  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
	  hipFree(dev_c);
  
  }
  
  	
  
  __global__ void convolute1_kernel(float *a, float *b, float *c, int filt_n, int filt_c, int filt_h, int filt_w,
	int st_h, int st_w, int pad, int in_c, int in_h, int in_w, int oh, int ow)
{
   int bid = blockIdx.x;
   int tid = threadIdx.x;
   int n1,c1,c2,h1,w1,h2,w2,start_h1,end_h1,start_w1,end_w1;
   float sum,elt1,elt2;
	 
   n1 = bid;
   c2 = tid;
   for(w2=0;w2<ow;w2++){
	   for(h2=0;h2<oh;h2++){
		   sum = 0.0;
		   start_h1 = st_h*h2-pad;
		   end_h1 = start_h1 + filt_h;
		   start_w1 = st_w*w2-pad;
		   end_w1 = start_w1 + filt_w;
		   for(c1=0;c1<in_c;c1++){
			   for(h1=start_h1;h1<end_h1;h1++){
				   for(w1=start_w1;w1<end_w1;w1++){
					   if(h1 >= 0 && h1 < in_h && w1 >= 0 && w1 < in_w){
						   elt1 = a[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)];
						   elt2 = b[IDX4C(c2,c1,h1-start_h1,w1-start_w1,filt_c,filt_h,filt_w)];
						   sum = sum + elt1*elt2;
					   }
				   }
			   }
		   }
		   c[IDX4C(n1,c2,h2,w2,filt_n,oh,ow)] = sum;   
	   }
   }
	   
}
 
/*
1st arg in_n of input tensor
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg filt_n of filter tensor
6th arg filt_c of filter tensor
7th arg filt_h of filter tensor
8th arg filt_w of filter tensor
9th arg input tensor
10th arg filter tensor
11th arg filter output-tensor
12th arg stride hight
13th arg stride width
14th arg padding   
*/

void cuda_convolute(int in_n, int in_c, int in_h, int in_w, int filt_n, int filt_c, int filt_h, int filt_w, float *a, float *b, float *c,
                int st_h, int st_w, int pad){
   int n1, n2, n3, oh, ow;
   float *dev_a, *dev_b, *dev_c;
 
   
   n1 = in_n * in_c * in_h * in_w;
   n2 = filt_n * filt_c * filt_h * filt_w;
   oh = (in_h+2*pad-filt_h)/st_h + 1;
   ow = (in_w+2*pad-filt_w)/st_w + 1;
   n3 = in_n * filt_n * oh * ow;  // n of filter generate n channel
   
   // Allocate for GPU
   CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_c, n3 * sizeof(float)));

 
   // copy from host a,b to GPU dev_a, dev_b
   CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
   CHECK(hipMemcpy(dev_b, b, n2 * sizeof(float), hipMemcpyHostToDevice));


   dim3 blocks(in_n,1,1);
   dim3 threads(filt_n,1,1);
   convolute1_kernel <<<blocks, threads>>>(dev_a, dev_b, dev_c, filt_n, filt_c, filt_h, filt_w, st_h, st_w, pad, in_c, in_h, in_w, oh, ow);
 
   // copy to host c from GPU dev_c
   CHECK(hipMemcpy(c, dev_c, n3 * sizeof(float), hipMemcpyDeviceToHost));

   // free 
   hipFree(dev_a);
   hipFree(dev_b);
   hipFree(dev_c);
   
}


  
__global__ void deconvolute1_kernel(float *a, float *b, float *c, int filt_n, int filt_c, int filt_h, int filt_w,
	int st_h, int st_w, int pad1, int pad, int in_c, int in_h, int in_w, int oh, int ow, int oh1, int ow1)
{
   int bid = blockIdx.x;
   int tid = threadIdx.x;
   int n1,c1,c2,h1,w1,h2,w2,start_h1,end_h1,start_w1,end_w1;
   float sum,elt1,elt2;
   
   n1 = bid;
   c2 = tid;
	   
   //full convolute. stride=1 always
   for(w2=0;w2<ow;w2++){
	   for(h2=0;h2<oh;h2++){
		   start_h1 = h2-pad1;  
		   end_h1 = start_h1 + filt_h;
		   start_w1 = w2-pad1;
		   end_w1 = start_w1 + filt_w;
		   sum = 0.0;
		   for(h1=start_h1;h1<end_h1;h1++){
			   for(w1=start_w1;w1<end_w1;w1++){
				   for(c1=0;c1<filt_n;c1++){        
					   if(h1 >= 0 && h1 < in_h && w1 >= 0 && w1 < in_w){
						   elt1 = a[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)]; //loss tensor
						   elt2 = b[IDX4C(c1,c2,h1-start_h1,w1-start_w1,filt_c,filt_h,filt_w)]; //filter tensor
						   sum = sum + elt1*elt2;
					   } 
				   }   
			   }
		   }
		   if(h2-pad >=0 && h2-pad < oh1 && w2-pad >= 0 && w2-pad < ow1){
			   c[IDX4C(n1,c2,h2-pad,w2-pad,filt_c,oh1,ow1)] = sum;
		   }             
	   }
   }
   
}
 
/*
1st arg in_n of input tensor
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg filt_n of filter tensor
6th arg filt_c of filter tensor
7th arg filt_h of filter tensor
8th arg filt_w of filter tensor
9th arg input loss tensor
10th arg filter tensor
11th arg output tensor
12th arg stride hight
13th arg stride width
14th arg padding   

memo
ex padding = 1
loss 4*4
filter 2*2
input 3*3  padding=1
(3-2+2*1)/1 + 1 = 4  
decovolute compute 5*5(3*3 padding=1) and save result range 3*3


*/
void cuda_deconvolute1(int in_n, int in_c, int in_h, int in_w, int filt_n, int filt_c, int filt_h, int filt_w,
                   float *a, float *b, float *c, int st_h, int st_w, int pad){
   int pad1, n1, n2, n3, oh, ow, oh1, ow1, i,j,k,l;
   float *b1;
   float *dev_a, *dev_b, *dev_c;
 
   
   n1 = in_n * in_c * in_h * in_w;
   n2 = filt_n * filt_c * filt_h * filt_w;
   pad1 = filt_h - 1;
   // pad1 = filt_h -1,  pad is original padding size
   oh = (in_h+2*pad1-filt_h)/st_h + 1;
   ow = (in_w+2*pad1-filt_w)/st_w + 1;
   oh1 = (in_h+2*(pad1-pad)-filt_h)/st_h + 1;
   ow1 = (in_w+2*(pad1-pad)-filt_w)/st_w + 1;
   n3 = in_n * filt_c * oh1 * ow1;  // channel of filter generate same channel input tensor
   b1 = (float *)malloc (n2 * sizeof (float));
 
	 
   //rotate 180 degree
   for(i=0;i<filt_n;i++){  
	   for(j=0;j<filt_c;j++){
		   for(k=0;k<filt_h;k++){
			   for(l=0;l<filt_w;l++){
				   b1[IDX4C(i,j,filt_h-k-1,filt_w-l-1,filt_c,filt_h,filt_w)] = b[IDX4C(i,j,k,l,filt_c,filt_h,filt_w)];
			   }
		   }
	   }
   }

   
   // Allocate for GPU
   CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_c, n3 * sizeof(float)));

 
   // copy from host a,b1,c to GPU dev_a, dev_b, dev_c
   CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
   CHECK(hipMemcpy(dev_b, b1, n2 * sizeof(float), hipMemcpyHostToDevice));
   CHECK(hipMemcpy(dev_c, c, n3 * sizeof(float), hipMemcpyHostToDevice));

   dim3 blocks(in_n,1,1);
   dim3 threads(filt_c,1,1);
   deconvolute1_kernel <<<blocks, threads>>>(dev_a, dev_b, dev_c, filt_n, filt_c, filt_h, filt_w, st_h, st_w, pad1, pad, in_c, in_h, in_w, oh, ow, oh1, ow1);
 
   // copy to host c from GPU dev_c
   CHECK(hipMemcpy(c, dev_c, n3 * sizeof(float), hipMemcpyDeviceToHost));

   // free 
   hipFree(dev_a);
   hipFree(dev_b);
   hipFree(dev_c);
   free(b1);
   
}




__global__ void deconvolute2_kernel(float *a1, float *a, float *b, float *c, int filt_n, int filt_c,int filt_h, int filt_w,
	int st_h, int st_w, int pad, int in_c, int in_h, int in_w, int loss_h, int loss_w)
{
   int bid = blockIdx.x;
   int tid = threadIdx.x;
   int n1,c1,c2,h1,w1,h2,w2,oh,ow,start_h1,end_h1,start_w1,end_w1;
   int j,k,l,k1,l1;
   float sum,elt1,elt2;
   
   n1 = bid;
   c2 = tid;
   // caution! stride=1 
   oh = (in_h+2*pad-filt_h) + 1;
   ow = (in_w+2*pad-filt_w) + 1;
   
   //dilate loss tensor.
   for(j=0;j<filt_n;j++){
	   for(k=0;k<loss_h;k++){
		   for(l=0;l<loss_w;l++){
			   elt1 = a[IDX4C(n1,j,k,l,in_c,loss_h,loss_w)];
			   k1 = st_h*k;
			   l1 = st_w*l;
			   a1[IDX4C(n1,j,k1,l1,in_c,in_h,in_w)] = elt1;
		   }
	   }
   }
   //full convulute. stride=1
   for(c2=0;c2<filt_c;c2++){
	   for(w2=0;w2<ow;w2++){
		   for(h2=0;h2<oh;h2++){
			   start_h1 = h2-pad;
			   end_h1 = start_h1 + filt_h;
			   start_w1 = w2-pad;
			   end_w1 = start_w1 + filt_w;
			   sum = 0.0;
			   for(h1=start_h1;h1<end_h1;h1++){
				   for(w1=start_w1;w1<end_w1;w1++){
					   for(c1=0;c1<filt_n;c1++){        
						   if(h1 >= 0 && h1 < in_h && w1 >= 0 && w1 < in_w){
							   elt1 = a1[IDX4C(n1,c1,h1,w1,in_c,in_h,in_w)]; //loss tensor
							   elt2 = b[IDX4C(c1,c2,h1-start_h1,w1-start_w1,filt_c,filt_h,filt_w)]; //filter tensor
							   sum = sum + elt1*elt2;
						   }
					   }
				   }   
			   }
			   c[IDX4C(n1,c2,h2,w2,filt_c,oh,ow)] = sum;              
		   }
	   }
   }
   
}




/*
dilate loss tensor 
e.g.

|1.0,2.0|
|3.0,4.0|

dilated stride=2
|1.0,0.0,2.0|
|0.0,0.0,0.0|
|3.0,0.0,4.0|


*/


/*
1st arg in_n of input loss tensor
2nd arg in_c of input loss tensor
3rd arg in_h of input loss  tensor
4th arg in_w of input loss tensor
5th arg filt_n of filter tensor
6th arg filt_c of filter tensor
7th arg filt_h of filter tensor
8th arg filt_w of filter tensor
9th arg input loss tensor
10th arg filter tensor
11th arg output tensor
12th arg stride hight
13th arg stride width
14th arg padding   
*/

void cuda_deconvolute2(int in_n, int in_c, int loss_h, int loss_w, int filt_n, int filt_c, int filt_h,int filt_w,
             float *a, float *b, float *c,int st_h, int st_w,int pad){
   int pad1, n1, n2, n3, oh, ow, i,j,k,l,in_h, in_w;
   float *a1, *b1;
   float *dev_a, *dev_a1, *dev_b, *dev_c;

 
	   
   // size for dilate
   in_h = loss_h + (loss_h - 1)*(st_h - 1);
   in_w = loss_w + (loss_w - 1)*(st_w - 1);

   n1 = in_n * in_c * in_h * in_w;  //loss tensor size 
   n2 = filt_n * filt_c * filt_h * filt_w;  //filter tensor size
   pad1 = (filt_h - 1) + pad;    //padding size with dilate
   oh = (in_h+2*pad1-filt_h) + 1; //output deconvolute tensor size. caution stride=1.
   ow = (in_w+2*pad1-filt_w) + 1; // 
   n3 = in_n * filt_c * oh * ow;   // 
   a1 = (float *)malloc (n1 * sizeof (float));
   b1 = (float *)malloc (n2 * sizeof (float));

   //rotate 180 degree
   for(i=0;i<filt_n;i++){  
	   for(j=0;j<filt_c;j++){
		   for(k=0;k<filt_h;k++){
			   for(l=0;l<filt_w;l++){
				   b1[IDX4C(i,j,filt_h-k-1,filt_w-l-1,filt_c,filt_h,filt_w)] = b[IDX4C(i,j,k,l,filt_c,filt_h,filt_w)];
			   }
		   }
	   }
   }


   // dilate 
   for(i=0;i<n1;i++){
	   a1[i] = 0.0;
   }

   CHECK(hipMalloc((void**)&dev_a1, n1 * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_a, in_n*1*loss_h*loss_w * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
   CHECK(hipMalloc((void**)&dev_c, n3 * sizeof(float)));

   CHECK(hipMemcpy(dev_a1, a1, n1 * sizeof(float), hipMemcpyHostToDevice));
   CHECK(hipMemcpy(dev_a, a, in_n*1*loss_h*loss_w  * sizeof(float), hipMemcpyHostToDevice));
   CHECK(hipMemcpy(dev_b, b1, n2 * sizeof(float), hipMemcpyHostToDevice));
   CHECK(hipMemcpy(dev_c, c, n3 * sizeof(float), hipMemcpyHostToDevice));

   dim3 blocks(in_n,1,1);
   dim3 threads(filt_c,1,1);
   deconvolute2_kernel <<<blocks, filt_c>> >(dev_a1, dev_a, dev_b, dev_c, filt_n, filt_c, filt_h, filt_w, st_h, st_w, pad1, in_c, in_h, in_w, loss_h, loss_w);
 
   // copy to host c from GPU dev_c
   CHECK(hipMemcpy(c, dev_c, n3 * sizeof(float), hipMemcpyDeviceToHost));
   
   // free 
   hipFree(dev_a);
   hipFree(dev_a1);
   hipFree(dev_b);
   hipFree(dev_c);
   free(a1);
   free(b1);
 
}

  
__global__ void gradfilter1_kernel(float *a, float *b, float *c, int filt_n, int filt_c, int filt_h, int filt_w, int loss_c, int loss_h, int loss_w, int st_h, int st_w, int pad, int in_c, int in_h, int in_w, int n)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int n1,c1,c2,h1,w1,h2,w2,h3,w3;
    float sum,elt1,elt2;

    n1 = bid;
    c2 = tid;
    
    for(c1=0;c1<filt_c;c1++){
    //h1,w1 is index of filter
        for(h1=0;h1<filt_h;h1++){
            for(w1=0;w1<filt_w;w1++){
                //h2,w2 is index of loss tensor
                sum = 0.0;
                for(h2=0;h2<loss_h;h2++){
                    for(w2=0;w2<loss_w;w2++){
                        //h3,w3 is index of input tensor
                        h3 = h1 - pad + h2;
                        w3 = w1 - pad + w2;
                        if(h3>=0 && h3<in_h && w3>=0 && w3<in_w){
                            elt1 = a[IDX4C(n1,c1,h3,w3,in_c,in_h,in_w)];    //input tensor
                            elt2 = b[IDX4C(n1,c2,h2,w2,loss_c,loss_h,loss_w)]; //loss tensor
                            sum = sum + elt1*elt2;
                        }
                    }
                }
                //set filter tensor
                c[IDX5C(n1,c2,c1,h1,w1,filt_n,filt_c,filt_h,filt_w)] =  sum;
            }
        }
    } 
               
}



  
/*
1st arg in_n of input tensor
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg filt_n of filter tensor
6th arg filt_c of filter tensor
7th arg filt_h of filter tensor
8th arg filt_w of filter tensor
9th arg loss_c of loss tensor
10th arg loss_h of loss tensor
11th arg loss_w of loss tensor
12th arg filter tensor
13th arg loss tensor
14th arg output tensor
15th arg stride hight
16th arg stride width
17th arg padding   
*/

void cuda_gradfilter1(int in_n, int in_c, int in_h, int in_w, int filt_n, int filt_c, int filt_h, int filt_w,
                 int loss_c, int loss_h, int loss_w, float *a, float *b, float *d, int st_h, int st_w, int pad){
    int n1,n2,n3,n4,i,j,k,l,m;
    float *c;
    float *dev_a, *dev_b, *dev_c;
    float elt;
  
    
    n1 = in_n * in_c * in_h * in_w;
    n2 = in_n * loss_c * loss_h * loss_w;
    n3 = in_n * filt_n * filt_c * filt_h * filt_w;
    n4 = filt_n * filt_c * filt_h * filt_w;
    c = (float *)malloc (n3 * sizeof (float));

    //initialize c
    for(i=0;i<n3;i++){
        c[i] = 0.0;
    }
  
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n3 * sizeof(float)));

    
    // copy from host a,b,c to GPU dev_a, dev_b, dev_c
    CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n2 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_c, c, n3 * sizeof(float), hipMemcpyHostToDevice));

    dim3 blocks(in_n,1,1);
    dim3 threads(filt_n,1,1);
    gradfilter1_kernel <<<blocks, threads>>>(dev_a, dev_b, dev_c, filt_n, filt_c, filt_h, filt_w, loss_c, loss_h, loss_w, st_h, st_w, pad, in_c, in_h, in_w, in_n);
  
    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(c, dev_c, n3 * sizeof(float), hipMemcpyDeviceToHost));

    //average
    // clear d
    for(i=0;i<n4;i++){
        d[i] = 0.0;
    }
    // copy from c to d and compute sum
    for(i=0;i<in_n;i++){
        for(j=0;j<filt_n;j++){
            for(k=0;k<filt_c;k++){
                for(l=0;l<filt_h;l++){
                    for(m=0;m<filt_w;m++){
                        elt = c[IDX5C(i,j,k,l,m,filt_n,filt_c,filt_h,filt_w)];
                        d[IDX4C(j,k,l,m,filt_c,filt_h,filt_w)] = d[IDX4C(j,k,l,m,filt_c,filt_h,filt_w)] + elt;
                    }
                }
            }
        }
    }
    // average
    for(i=0;i<n4;i++){
        d[i] = d[i] / (float)in_n;
    }
    
    
    // free 
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    free(c);
}

  
__global__ void gradfilter2_kernel(float *a, float *b1, float *b, float *c, int filt_n, int filt_c, int filt_h, int filt_w, int loss_c, int loss_h, int loss_w, int st_h, int st_w, int pad, int in_c, int in_h, int in_w, int n)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int n1,c1,c2,h1,w1,h2,w2,h3,w3,loss_h1,loss_w1,j,k,l,k1,l1;
    float sum,elt1,elt2;
    
    n1 = bid;
    c2 = tid;
    //dilated loss tensor size
    loss_h1 = loss_h+(loss_h-1)*(st_h-1);
    loss_w1 = loss_w+(loss_w-1)*(st_w-1);
    //dilate loss tensor.
    for(j=0;j<loss_c;j++){
        for(k=0;k<loss_h;k++){
            for(l=0;l<loss_w;l++){
                elt1 = b[IDX4C(n1,j,k,l,loss_c,loss_h,loss_w)];
                k1 = st_h*k;
                l1 = st_w*l;
                b1[IDX4C(n1,j,k1,l1,loss_c,loss_h1,loss_w1)] = elt1;
            }
        }
    }
    //convolute input tensor with dilated loss tensor. cuation stride is always 1. 
    for(c1=0;c1<filt_c;c1++){
    //h1,w1 is index of filter
        for(h1=0;h1<filt_h;h1++){
            for(w1=0;w1<filt_w;w1++){
                //h2,w2 is index of loss tensor
                sum = 0.0;
                for(h2=0;h2<loss_h1;h2++){
                    for(w2=0;w2<loss_w1;w2++){
                        //h3,w3 is index of input tensor
                        h3 = h1 - pad + h2;
                        w3 = w1 - pad + w2;
                        if(h3>=0 && h3<in_h && w3>=0 && w3<in_w){
                            elt1 = a[IDX4C(n1,c1,h3,w3,in_c,in_h,in_w)];    //input tensor
                            elt2 = b1[IDX4C(n1,c2,h2,w2,loss_c,loss_h1,loss_w1)]; //loss tensor
                            sum = sum + elt1*elt2;
                        }
                    }
                }
                //set filter tensor
                c[IDX5C(n1,c2,c1,h1,w1,filt_n,filt_c,filt_h,filt_w)] = + sum;
            }
        }
    } 
        
}

/*
dilate loss tensor 
e.g.

|1.0,2.0|
|3.0,4.0|

dilated stride=2
|1.0,0.0,2.0|
|0.0,0.0,0.0|
|3.0,0.0,4.0|


*/

/*
gradfilter2 is for stride >= 2. This one requires dilate
1st arg in_n of input tensor
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg filt_n of filter tensor
6th arg filt_c of filter tensor
7th arg filt_h of filter tensor
8th arg filt_w of filter tensor
9th arg loss_c of loss tensor
10th arg loss_h of loss tensor
11th arg loss_w of loss tensor
12th arg filter tensor
13th arg loss tensor
14th arg output tensor
15th arg stride hight
16th arg stride width
17th arg padding  

*/
void cuda_gradfilter2(int in_n, int in_c, int in_h, int in_w, int filt_n, int filt_c, int filt_h, int filt_w, 
                 int loss_c, int loss_h, int loss_w, float *a, float *b, float *d, int st_h, int st_w, int pad){
    int n1,n2,n3,n4,n5,i,j,k,l,m;
    float *b1,*c;
    float *dev_a, *dev_b, *dev_b1, *dev_c;
    float elt;
  
    n1 = in_n * in_c * in_h * in_w;
    n2 = in_n * loss_c * loss_h * loss_w;
    n3 = in_n * filt_n * filt_c * filt_h * filt_w;
    n4 = filt_n * filt_c * filt_h * filt_w;
    n5 = in_n * loss_c * (loss_h+(loss_h-1)*(st_h-1)) * (loss_w+(loss_w-1)*(st_w-1));  // dilated loss tensor size  
    b1 = (float *)malloc (n5 * sizeof (float));  // dilate loss tensor area
    c = (float *)malloc (n3 * sizeof (float));

    //initialize c
    for(i=0;i<n3;i++){
        c[i] = 0.0;
    }
    //initialize b1
    for(i=0;i<n5;i++){
        b1[i] = 0.0;
    }
  
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n2 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b1, n5 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n3 * sizeof(float)));

    
    // copy from host a,b,c to GPU dev_a, dev_b, dev_c
    CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n2 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b1, b1, n5 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_c, c, n3 * sizeof(float), hipMemcpyHostToDevice));

    dim3 blocks(in_n,1,1);
    dim3 threads(filt_n,1,1);
    gradfilter2_kernel <<<blocks, threads>> >(dev_a, dev_b1, dev_b, dev_c, filt_n, filt_c, filt_h, filt_w, loss_c, loss_h, loss_w, st_h, st_w, pad, in_c, in_h, in_w, in_n);
  
    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(c, dev_c, n3 * sizeof(float), hipMemcpyDeviceToHost));

    //average
    // clear d
    for(i=0;i<n4;i++){
        d[i] = 0.0;
    }
    // copy from c to d and compute sum
    for(i=0;i<in_n;i++){
        for(j=0;j<filt_n;j++){
            for(k=0;k<filt_c;k++){
                for(l=0;l<filt_h;l++){
                    for(m=0;m<filt_w;m++){
                        elt = c[IDX5C(i,j,k,l,m,filt_n,filt_c,filt_h,filt_w)];
                        d[IDX4C(j,k,l,m,filt_c,filt_h,filt_w)] = d[IDX4C(j,k,l,m,filt_c,filt_h,filt_w)] + elt;
                    }
                }
            }
        }
    }
    // average
    for(i=0;i<n4;i++){
        d[i] = d[i] / (float)in_n;
    }
     
    
    // free 
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_b1);
    hipFree(dev_c);
    free(b1);
    free(c);
}



__global__ void full_kernel(float *a, float *b, int in_n, int in_c, int in_h, int in_w, int n)
{
    int tid = threadIdx.x;
    int n1,i,j,k;
    float elt;
    if(tid < n)
    {   
        n1 = tid;
        for(i=0;i<in_c;i++){
            for(j=0;j<in_h;j++){
                for(k=0;k<in_w;k++){
                    elt = a[IDX4C(n1,i,j,k,in_c,in_h,in_w)];
                    b[IDX2C(n1,i*in_h*in_w + j*in_w + k,in_n)] = elt;
                }
            }
        }
    }
}
  
/*
1st arg in_n of input tensor 4DIM
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg input tensor
6th arg output tensor
*/

void cuda_full(int in_n, int in_c, int in_h, int in_w, float *a, float *b){
    int n1,n;
    float *dev_a, *dev_b;
 
    n1 = in_n * in_c * in_h * in_w;
    n = in_n;
      
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n1 * sizeof(float)));
  
    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n1 * sizeof(float), hipMemcpyHostToDevice));

    full_kernel << <1, n>> >(dev_a, dev_b, in_n, in_c, in_h, in_w, n);
  
    // copy to host d from GPU dev_d
    CHECK(hipMemcpy(b, dev_b, n1 * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
    hipFree(dev_b);

}


__global__ void unfull_kernel(float *a, float *b, int in_n, int in_c, int in_h, int in_w, int n)
{
    int tid = threadIdx.x;
    int n1,i,j,k;
    float elt;
    if(tid < n)
    {   
        n1 = tid;
        for(i=0;i<in_c;i++){
            for(j=0;j<in_h;j++){
                for(k=0;k<in_w;k++){
                    elt = a[IDX2C(n1,i*in_h*in_w + j*in_w + k,in_n)];
                    b[IDX4C(n1,i,j,k,in_c,in_h,in_w)] = elt;
                }
            }
        }
    }
}
  
/*
1st arg in_n of input tensor 4DIM
2nd arg in_c of input tensor
3rd arg in_h of input tensor
4th arg in_w of input tensor
5th arg input tensor
6th arg output tensor
*/

void cuda_unfull(int in_n, int in_c, int in_h, int in_w, float *a, float *b){
    int n1,n;
    float *dev_a, *dev_b;
    
    n1 = in_n * in_c * in_h * in_w;
    n = in_n;
      
      // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n1 * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n1 * sizeof(float)));
  
    // copy from host a,b1,c to GPU dev_a, dev_b, dev_c
    CHECK(hipMemcpy(dev_a, a, n1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n1 * sizeof(float), hipMemcpyHostToDevice));

    unfull_kernel << <1, n>> >(dev_a, dev_b, in_n, in_c, in_h, in_w, n);
  
    // copy to host d from GPU dev_d
    CHECK(hipMemcpy(b, dev_b, n1 * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
    hipFree(dev_b);

}

__global__ void emult1_kernel(float *a, float *b, float *c, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n)
    {
        c[tid] = a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
}


void cuda_emult(int n, float *a, float *b,float *c){
    float *dev_a, *dev_b, *dev_c;

    
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

    emult1_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}




__global__ void sigmoid_kernel(float *a, float *b, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n)
    {   
        b[tid] = SIGMOID(a[tid]);
        tid += blockDim.x * gridDim.x;
    }
}

void cuda_activate_sigmoid(int n, float *a, float *b){
    float *dev_a, *dev_b;

   
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

    sigmoid_kernel << <128, 128 >> >(dev_a, dev_b, n);

    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(b, dev_b, n * sizeof(float), hipMemcpyDeviceToHost));
    
    // free 
    hipFree(dev_a);
    hipFree(dev_b);

}

  
__global__ void tanh_kernel(float *a, float *b, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n)
    {
        b[tid] = tanh(a[tid]);
        tid += blockDim.x * gridDim.x;
    }
}


void cuda_activate_tanh(int n, float *a, float *b){
    float *dev_a, *dev_b;

    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

    tanh_kernel << <128, 128 >> >(dev_a, dev_b, n);

    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(b, dev_b, n * sizeof(float), hipMemcpyDeviceToHost));
    
    // free 
    hipFree(dev_a);
    hipFree(dev_b);
}


  
__global__ void relu_kernel(float *a, float *b, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n)
    {   
        if(a[tid] >= 0)
            b[tid] = a[tid];
        else 
            b[tid] = 0.0;
        tid += blockDim.x * gridDim.x;
    }
}


void cuda_activate_relu(int n, float *a, float *b){
    float *dev_a, *dev_b;

    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

    relu_kernel << <128, 128 >> >(dev_a, dev_b, n);

    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(b, dev_b, n * sizeof(float), hipMemcpyDeviceToHost));
    
    // free 
    hipFree(dev_a);
    hipFree(dev_b);
}

void cuda_activate_softmax(int r1, int c1, float *a, float *b){
    int i, j, k;
    float max,sum,delta;

    
    //calculate softmax
    delta = 0.01;
    for(i=0;i<r1;i++){
        for(j=0;j<c1;j++){
            max = -3.402823e38;
            for(k=0;k<c1;k++){
                if(a[IDX2C(i,k,r1)] > max)
                    max = a[IDX2C(i,k,r1)];
            }
            sum = 0.0;
            for(k=0;k<c1;k++){
                sum = sum + exp(a[IDX2C(i,k,r1)] - max);
            }
            b[IDX2C(i,j,r1)] = exp(a[IDX2C(i,j,r1)] - max) / (sum+delta);
            
        }
    }

}



__global__ void differ_sigmoid_kernel(float *a, float *b, float *c, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n)
    {   
        
        c[tid] = a[tid] * ((1 - SIGMOID(b[tid])) * SIGMOID(b[tid]));
        tid += blockDim.x * gridDim.x;
    }
}


void cuda_differ_sigmoid(int n, float *a, float *b, float *c){
    float *dev_a, *dev_b, *dev_c;

    
    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

    differ_sigmoid_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

}


__global__ void differ_tanh_kernel(float *a, float *b, float *c, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n)
    {   
        c[tid] = a[tid] * (1/(cosh(b[tid]) * cosh(b[tid])));
        tid += blockDim.x * gridDim.x;
    }
}


void cuda_differ_tanh(int n, float *a, float *b, float *c){
    float *dev_a, *dev_b, *dev_c;

    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

    differ_tanh_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}



__global__ void differ_relu_kernel(float *a, float *b, float *c, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n)
    {   
        if(b[tid] >= 0)
            c[tid] = a[tid];
        else 
            c[tid] = 0.0;
        tid += blockDim.x * gridDim.x;
    }
}


void cuda_differ_relu(int n, float *a, float *b, float *c){
    float *dev_a, *dev_b, *dev_c;

    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

    differ_relu_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, n);

    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));

    // free 
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

}



void add_diff1(int r1, int c1, float *a, float *b, int x, int y, float val) {
    int i, j;
    
    for(i=0;i<r1;i++){
        for(j=0;j<c1;j++){
            if(i==x && j==y)
                b[IDX2C(i,j,r1)] = a[IDX2C(i,j,r1)] + (float)val;
            else 
                b[IDX2C(i,j,r1)] = a[IDX2C(i,j,r1)];
        }
    }

}


void add_diff2(int n1, int c1, int h1, int w1, float *a, float *b, int n2, int c2, int h2, int w2,float val){
    int i, j, k, l;
    
    for(i=0;i<n1;i++){
        for(j=0;j<c1;j++){
            for(k=0;k<h1;k++){
                for(l=0;l<w1;l++){
                    if(i==n2 && j==c2 && k==h2 && l==w2){
                        b[IDX4C(i,j,k,l,c1,h1,w1)] = a[IDX4C(i,j,k,l,c1,h1,w1)] + (float)val;
                    }
                    else {
                        b[IDX4C(i,j,k,l,c1,h1,w1)] = a[IDX4C(i,j,k,l,c1,h1,w1)];
                    }
                }
            }
        }
    }


}




__global__ void dropout1_kernel(float *a, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n)
    {
        a[tid] = 1.0;
        tid += blockDim.x * gridDim.x;
    }
}

/*
1st arg size of mask tensor
2nd arg rate of dropout

return mask tensor
element of mask tensor is basicaly 1.0.
element of dropout rate is 0.0.
when forward and backward, generate Hadamard product with mask tensor
*/

void cuda_dropout(int n, float dropout_rate, float *a){ 
    int count,i,j;
    float *dev_a;


    // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));

    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));

    dropout1_kernel << <128, 128 >> >(dev_a, n);

    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(a, dev_a, n * sizeof(float), hipMemcpyDeviceToHost));


    // dropout
    count = (int)(double(n)*dropout_rate);
    for(i=0;i<count;i++){
        j = rand() % n;
        a[j] = 0.0;
    }

    // free 
    hipFree(dev_a);
}


__global__ void sgd1_kernel(float *a, float *b, float *c, float lr, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n)
    {
        c[tid] = a[tid] - b[tid]*lr;
        tid += blockDim.x * gridDim.x;
    }
}
/*
w - g*lr
w is weight matrix.
g is gradient matrix.
when element of w is zero result is zero. This means dropout.
return updated weight matrix.

1st arg is size of vectorized matrix
2nd arg is weight matrix or tensor
3rd arg is gradient matrix or tensor
4th arg is output tensor
5th arg is learning rate
*/

void cuda_sgd(int n, float *a, float *b, float *c, float lr){
    float *dev_a, *dev_b, *dev_c;


        // Allocate for GPU
    CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
    CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));


    // copy from host a,b to GPU dev_a, dev_b
    CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));

    sgd1_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, lr, n);

    // copy to host c from GPU dev_c
    CHECK(hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost));


    // free 
    hipFree(dev_a);
    hipFree(dev_b);    
    hipFree(dev_c);

}

  
  /*
	def momentum(v, g, lr) do
	  Matrex.apply(v, g, fn v, g -> 0.5 * v - lr * g end)
	end
  */
  __global__ void momentum_kernel(float *a, float *b, float *c, float *d, float *e, float lr, int n)
  {
	  int tid = threadIdx.x + blockIdx.x * blockDim.x;
	  while (tid < n)
	  {   
		  
		  d[tid] = ((0.9 * b[tid]) - (lr * c[tid]));
		  e[tid] = a[tid] + d[tid];
		  
		  tid += blockDim.x * gridDim.x;
	  }
  }
  
  /*
  1st arg row-size of vectorized each-matrix
  2nd arg wight-matrix    (a)
  3rd arg v-matrix        (b)
  4th arg gradient-matrix (c)
  5th arg output next_v-matrix (d)
  6th arg output weight_matrix (e)
  7th arg learning rate
  
  */
  
  void cuda_momentum(int n, float *a, float *b, float *c, float *d, float *e, float lr){
	  float *dev_a, *dev_b, *dev_c ,*dev_d, *dev_e;
	
	 
	  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_d, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_e, n * sizeof(float)));
	
	  // copy from host a,b to GPU dev_a, dev_b
	  CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_c, c, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_d, d, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_e, e, n * sizeof(float), hipMemcpyHostToDevice));
	
	  momentum_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, dev_d, dev_e, lr, n);
	
	  // copy to host d from GPU dev_d
	  CHECK(hipMemcpy(d, dev_d, n * sizeof(float), hipMemcpyDeviceToHost));
	  CHECK(hipMemcpy(e, dev_e, n * sizeof(float), hipMemcpyDeviceToHost));
  
	  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
	  hipFree(dev_c);
	  hipFree(dev_d);
	  hipFree(dev_e);
	  
  }
  

  /* ADAGRAD
	  h1 = h + grad*grad
	  lr1 = lr/(sqrt(h1))
	  w1 = w - lr1 * grad 
  
	  a[] = w
	  b[] = h
	  c[] = grad
	  d[] = h1
	  e[] = w1
  */
	
  __global__ void adagrad_kernel(float *a, float *b, float *c, float *d, float *e, float lr, int n)
  {
	  int tid = threadIdx.x + blockIdx.x * blockDim.x;
	  float lr1;
	  while (tid < n)
	  {   
		  d[tid] = b[tid] + c[tid]*c[tid];
		  if(d[tid] != 0.0)
			  lr1 = lr/(sqrt(d[tid]));
		  else
			  lr1 = lr;
		  e[tid] = a[tid] - lr1 * c[tid];
  
		  tid += blockDim.x * gridDim.x;
	  }
  }
   
  /*
  1st arg row-size of vectorized each-matrix
  2nd arg wight-matrix (a)
  3rd arg h-matrix     (b)
  4th arg grad-matrix  (c)
  5th arg output new-h (d)
  6th arg output new-w (e)
  7th arg learning rate
  */
  
  void cuda_adagrad(int n, float *a, float *b, float *c, float *d, float *e, float lr){
	  float *dev_a, *dev_b, *dev_c, *dev_d, *dev_e;
	  
	  
	  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_d, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_e, n * sizeof(float)));
	
	  // copy from host a,b to GPU dev_a, dev_b
	  CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_c, c, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_d, d, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_e, e, n * sizeof(float), hipMemcpyHostToDevice));
	
	  adagrad_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, dev_d, dev_e, lr, n);
	
	  // copy to host d,e from GPU dev_d,dev_e
	  CHECK(hipMemcpy(d, dev_d, n * sizeof(float), hipMemcpyDeviceToHost));
	  CHECK(hipMemcpy(e, dev_e, n * sizeof(float), hipMemcpyDeviceToHost));
  
	  
  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
	  hipFree(dev_c);
	  hipFree(dev_d);
	  hipFree(dev_e);
	  
  }
  


  /* RMSprop
	  h1 = alpha * h + (1 - alpha) * grad*grad
	  lr1 = lr /(sqrt(h) + epsilon)
	  w1 = w - lr1 * grad 
  
	  a[] = w
	  b[] = h
	  c[] = grad
	  d[] = h1
	  e[] = w1
  */
	
  __global__ void rms_kernel(float *a, float *b, float *c, float *d, float *e, float lr, int n)
  {
	  int tid = threadIdx.x + blockIdx.x * blockDim.x;
	  float lr1,alpha,epsilon;
	  alpha = 0.99;
	  epsilon = 10.0e-7;
	  while (tid < n)
	  {   
		  d[tid] = alpha * b[tid] + (1-alpha)*c[tid]*c[tid];
		  lr1 = lr/(sqrt(d[tid])+epsilon);
		  e[tid] = a[tid] - lr1*c[tid];
  
		  tid += blockDim.x * gridDim.x;
	  }
  }
   
  /*
  1st arg row-size of vectorized each-matrix
  2nd arg wight-matrix (a)
  3rd arg h-matrix     (b)
  4th arg grad-matrix  (c)
  5th arg output new-h (d)
  6th arg output new-w (e)
  7th arg learning rate
  */
  
  void cuda_rms(int n, float *a, float *b, float *c, float *d, float *e, float  lr){
	  float *dev_a, *dev_b, *dev_c, *dev_d, *dev_e;
	  
	 
	  // Allocate for GPU
	  CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_d, n * sizeof(float)));
	  CHECK(hipMalloc((void**)&dev_e, n * sizeof(float)));
	
	  // copy from host a,b to GPU dev_a, dev_b
	  CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_c, c, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_d, d, n * sizeof(float), hipMemcpyHostToDevice));
	  CHECK(hipMemcpy(dev_e, e, n * sizeof(float), hipMemcpyHostToDevice));
	
	  rms_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, dev_d, dev_e, lr, n);
	
	  // copy to host d,e from GPU dev_d,dev_e
	  CHECK(hipMemcpy(d, dev_d, n * sizeof(float), hipMemcpyDeviceToHost));
	  CHECK(hipMemcpy(e, dev_e, n * sizeof(float), hipMemcpyDeviceToHost));
  
	  
  
	  // free 
	  hipFree(dev_a);
	  hipFree(dev_b);
	  hipFree(dev_c);
	  hipFree(dev_d);
	  hipFree(dev_e);
	  
  }
  


  /* ADAM
	  beta1 = 0.9
	  beta2 = 0.999
	  epsilon = 10.0e-7
	  alpha = 0.001
	  m1 = beta1 * m + (1 - beta1) * grad
	  v1 = beta2 * v + (1 - beta2) * grad^2
	  m2 = m1/(1 - beta1)
	  v2 = v1/(1 - beta2)
	  w1 = w - alpha * m2/(sqrt(v2)+epsilon)
  
	  a[] is w
	  b[] is m
	  c[] is v
	  d[] is grad
	  e[] is m1
	  f[] is v1
	  g[] is w1
   */
	
   __global__ void adam_kernel(float *a, float *b, float *c, float *d, float *e, float *f, float *g, float lr, int n)
   {
       int tid = threadIdx.x + blockIdx.x * blockDim.x;
       float beta1,beta2,epsilon,m2,v2;
       beta1 = 0.9;
       beta2 = 0.999;
       epsilon = 10.0e-7;
       //alpha = 0.001;
   
       while (tid < n){   
           e[tid] = beta1 * b[tid] + (1 - beta1) * d[tid];
           f[tid] = beta2 * c[tid] + (1 - beta2) * d[tid]*d[tid];
           m2 = e[tid]/(1-beta1);
           v2 = f[tid]/(1-beta2);
           g[tid] = a[tid] - lr * (m2/(sqrt(v2)+epsilon));
            
           tid += blockDim.x * gridDim.x;
       }
   }
      
   /*
   1st arg row-size of vectorized each-matrix
   2nd arg w-matrix     (a)
   3rd arg m-matrix     (b)
   4th arg v-matrix     (c)
   5th arg grad-matrix  (d)
   6th arg output m1    (e)
   7th arg output v1    (f)
   8th arg output w1    (g)
   9th arg learning rate
   
   */
   void cuda_adam(int n, float *a, float *b, float *c, float *d, float *e, float *f, float *g, float lr){
       float *dev_a, *dev_b, *dev_c, *dev_d, *dev_e, *dev_f, *dev_g;
       
       // Allocate for GPU
       CHECK(hipMalloc((void**)&dev_a, n * sizeof(float)));
       CHECK(hipMalloc((void**)&dev_b, n * sizeof(float)));
       CHECK(hipMalloc((void**)&dev_c, n * sizeof(float)));
       CHECK(hipMalloc((void**)&dev_d, n * sizeof(float)));
       CHECK(hipMalloc((void**)&dev_e, n * sizeof(float)));
       CHECK(hipMalloc((void**)&dev_f, n * sizeof(float)));
       CHECK(hipMalloc((void**)&dev_g, n * sizeof(float)));
   
       // copy from host a,b to GPU dev_a, dev_b
       CHECK(hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice));
       CHECK(hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice));
       CHECK(hipMemcpy(dev_c, c, n * sizeof(float), hipMemcpyHostToDevice));
       CHECK(hipMemcpy(dev_d, d, n * sizeof(float), hipMemcpyHostToDevice));
       CHECK(hipMemcpy(dev_e, e, n * sizeof(float), hipMemcpyHostToDevice));
       CHECK(hipMemcpy(dev_f, f, n * sizeof(float), hipMemcpyHostToDevice));
       CHECK(hipMemcpy(dev_g, g, n * sizeof(float), hipMemcpyHostToDevice));
       
       adam_kernel << <128, 128 >> >(dev_a, dev_b, dev_c, dev_d, dev_e, dev_f, dev_g, lr, n);
       
       // copy to host d,e from GPU dev_d,dev_e
       CHECK(hipMemcpy(e, dev_e, n * sizeof(float), hipMemcpyDeviceToHost));
       CHECK(hipMemcpy(f, dev_f, n * sizeof(float), hipMemcpyDeviceToHost));
       CHECK(hipMemcpy(g, dev_g, n * sizeof(float), hipMemcpyDeviceToHost));
         
     
       // free 
       hipFree(dev_a);
       hipFree(dev_b);
       hipFree(dev_c);
       hipFree(dev_d);
       hipFree(dev_e);
       hipFree(dev_f);
       hipFree(dev_g);
       
       
   }
     
     
   
  